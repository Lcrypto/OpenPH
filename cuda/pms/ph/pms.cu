#include "hip/hip_runtime.h"
inline void pms(int *d_rows_mp, int *d_aux_mp, int *d_low, int *d_arglow, int *d_dims, int *d_dims_order, const int m, const int p, int complex_dimension, int *d_beta, float *resRecord, float *timeRecord, int *p_iter, dim3 NBm, dim3 TPBm){

    // Auxiliary variables
    int *d_aux;
    int *d_clear;
    hipMalloc((void**)&d_aux, m * sizeof(int));
    hipMalloc((void**)&d_clear, m * sizeof(int));

    // -----------------------
    // Do some pre-processing work
    // -----------------------

    // d_classes
    int *d_classes;
    hipMalloc((void**)&d_classes, m * sizeof(int));
    fill<<<NBm, TPBm>>>(d_classes, 0, m);

    // Compute simplex dimensions (on device)
    // Get maximum dimension (TODO: Change Twist code too)
    int cdim = complex_dimension + 2; // -1, 0, 1, 2, ..., complex_dim
    int *d_aux_cdim;    // Auxiliary vector of size cdim 
    hipMalloc((void**)&d_aux_cdim, cdim * sizeof(int));
    printf("passed compute_dimension_order!!\n");
    printf("cdim = %d\n", cdim);

    // locks
    int *d_locks_cdim;
    hipMalloc((void**)&d_locks_cdim, cdim * sizeof(int));

    // updated

    // -----------------------
    // Phase 0
    // -----------------------

    alpha_beta_reduce<<<NBm, TPBm>>>(d_low, d_beta, d_classes, 
            d_rows_mp, d_arglow, m, p);

    int converged = is_reduced(d_aux, d_low, m, NBm, TPBm);
    printf("converged %d\n", converged);

    while (! converged ){

        // -----------------------
        // Main iteration : Phase I 
        // -----------------------

        // TODO: In get_ceilings, atomicCAS needs pointer in first
        // argument. Check if this is being given correctly
        // TODO: atomicMAX?
        fill<<<NBm, TPBm>>>(d_aux, 0, m);
        fill<<<NBm, TPBm>>>(d_locks_cdim, 0, cdim);
        fill<<<NBm, TPBm>>>(d_aux_cdim, 0, cdim);
        phase_i<<<NBm, TPBm>>>(d_dims, d_dims_order, 
                d_low, d_arglow, d_classes, d_clear,   
                d_aux, d_aux_cdim, d_locks_cdim, m);

        // -----------------------
        // Main iteration : Phase II 
        // -----------------------

        phase_ii<<<NBm, TPBm>>>(d_low, d_beta, d_classes, 
                d_arglow, d_rows_mp, d_aux_mp, m, p);

        // record iteration
        //record_iteration();

        // Check again if its reduced
        converged = is_reduced(d_aux, d_low, m, NBm, TPBm);
    }

    set_unmarked<<<NBm, TPBm>>>(d_classes, d_low, d_arglow, 
            d_rows_mp, m, p);

    hipFree(d_locks_cdim);
    hipFree(d_aux_cdim);
    hipFree(d_aux);
    hipFree(d_clear);
    hipFree(d_classes);

}

