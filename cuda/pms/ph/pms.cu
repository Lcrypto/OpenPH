inline void pms(int *d_rows_mp, int *d_aux_mp, int *d_low, int *d_arglow, int *d_dims, int *d_dims_order, int *d_dims_order_next, int *d_dims_order_start, const int m, const int p, int complex_dimension, int *d_left, int *d_beta, float *resRecord, float *timeRecord, int *p_iter, dim3 NBm, dim3 TPBm, dim3 NBcdim, dim3 TPBcdim){
    //  d_pivots[j] = 1  <=> d_arglow[d_low[j]] = j 

    // Auxiliary variables
    int *d_aux;
    int *d_clear;
    hipMalloc((void**)&d_aux, m * sizeof(int));
    hipMalloc((void**)&d_clear, m * sizeof(int));

    // -----------------------
    // Do some pre-processing work
    // -----------------------

    // d_classes
    int *d_classes;
    hipMalloc((void**)&d_classes, m * sizeof(int));
    fill<<<NBm, TPBm>>>(d_classes, 0, m);
    hipDeviceSynchronize();

    // Compute simplex dimensions (on device)
    // Get maximum dimension 
    int cdim = complex_dimension + 2; // -1, 0, 1, 2, ..., complex_dim
    int *d_aux_cdim;    // Auxiliary vector of size cdim 
    hipMalloc((void**)&d_aux_cdim, cdim * sizeof(int));
    printf("cdim = %d\n", cdim);

    // -----------------------
    // Phase 0
    // -----------------------

    // Mark pivots and clear corresponding positives
    mark_pivots_and_clear<<<NBm, TPBm>>>(d_low, d_beta, 
            d_classes, d_rows_mp, d_arglow, m, p);
    hipDeviceSynchronize();

    int converged = is_reduced(d_aux, d_low, m, NBm, TPBm);

    int iter = 0;
    thrust::device_ptr<int> d_classes_ptr = thrust::device_pointer_cast(d_classes);
    //int num_zeros = thrust::count(d_classes_ptr, d_classes_ptr + m, 0);
    //printf("num_zeros=%d\n", num_zeros);

    while (! converged ){

        printf("iter=%d\n", iter);

        // -----------------------
        // Main iteration : Phase I 
        // -----------------------

        fill<<<NBm, TPBm>>>(d_aux, 0, m);
        fill<<<NBm, TPBm>>>(d_aux_cdim, -1, cdim); // d_ceil
        fill<<<NBm, TPBm>>>(d_clear, 0, m);
        hipDeviceSynchronize();

        transverse_dimensions<<<NBcdim, TPBcdim>>>(d_dims, d_dims_order,
                d_dims_order_next, d_dims_order_start, 
                d_low, d_arglow, d_classes, d_clear,   
                d_aux, d_aux_cdim, cdim);
        hipDeviceSynchronize();

        clear_phase_i<<<NBm, TPBm>>>(d_low, d_classes, d_rows_mp, d_clear, m, p);
        hipDeviceSynchronize();

        // -----------------------
        // Main iteration : Phase II 
        // -----------------------

        phase_ii<<<NBm, TPBm>>>(d_low, d_left, d_classes, 
                d_arglow, d_rows_mp, d_aux_mp, m, p);
        hipDeviceSynchronize();

        // Check again if its reduced
        converged = is_reduced(d_aux, d_low, m, NBm, TPBm);

        // iter
        iter++;

        //num_zeros = thrust::count(d_classes_ptr, d_classes_ptr + m, 0);

        // record iteration
        //record_iteration();

    }

    set_unmarked<<<NBm, TPBm>>>(d_classes, d_low, d_arglow, d_rows_mp, m, p);

    hipFree(d_aux_cdim);
    hipFree(d_aux);
    hipFree(d_clear);
    hipFree(d_classes);

}

