#include "hip/hip_runtime.h"

__device__ int d_is_reduced = 1;
void __global__ matrix_is_reduced(int *d_lows, int *d_aux, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int low_j = d_lows[tid];
        if (low_j > -1){
            atomicAdd(d_aux+low_j, 1);
            if (d_aux[low_j] > 1)
                d_is_reduced = 0;
        }
    }
} 

void __global__ compute_dims_order(int *d_dims, int *d_dims_order, int *d_last_pos, int m, int *d_sentinel){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int j = tid;
        // set lock
        //printf("{tid=%d, lock=%d}, ", tid, lock);
        do {} while(d_lock != j);
        // do stuff
        int dim_j = d_dims[tid];
        d_dims_order[tid] = d_last_pos[dim_j+1]+1;
        d_last_pos[dim_j+1] += 1;
        //printf("[j=%d, dim_j=%d, lock=%d, d_last_pos=%d], ", j, dim_j, lock, d_last_pos[dim_j+1]);
        // free lock
        d_lock = j+1;
        __syncthreads();
    }
}

void __global__ alpha_beta_reduce(int *d_low, int *d_beta, int *d_classes, int *d_rows_mp, int *d_arglow, int m, int p){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int alpha = d_low[tid];
        int beta  = d_beta[tid];
        if (alpha == beta && beta > -1){
            // tid is "negative"
            d_classes[tid] = -1;
            int pos_pair = d_beta[tid];
            clear_column(pos_pair, d_rows_mp, p);
            d_arglow[pos_pair] = beta;
            d_low[pos_pair] = -1;
            d_classes[pos_pair] = 1;
            d_low[beta] = pos_pair;
        }
    }
}


void __global__ get_dims_order_start(int *d_dims, int *d_dims_order, int *d_dims_order_start, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_dims_order[tid] == 0){
            int cdim_pos = d_dims[tid] + 1;
            d_dims_order_start[cdim_pos] = tid;
        }
    }
}

void __global__ count_simplices_dim(int *d_dim_count, int *d_dims, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_dims[tid] > -1){
            atomicAdd(d_dim_count+d_dims[tid], 1);
        }
    }
}

void __global__ phase_i_cdim(int *d_dims, int *d_dims_order, int *d_dims_order_next, int *d_dims_order_start, int *d_low, int *d_arglow, int *d_classes, int *d_clear, int *d_visited, int *d_ceil_cdim, int *d_next_cdim, int m, int cdim){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < cdim){
        int iterate = 1;
        int dim_j; // -1, 0, 1, ..., complex_dim
        int cdim_pos;
        int dim_ceil; // initialized at -1 
        int low_j;
        int j = d_dims_order_start[tid];
        while (iterate){
            dim_j = d_dims[j];
            cdim_pos = dim_j + 1; 
            dim_ceil = d_ceil_cdim[cdim_pos];
            low_j = d_low[j];
            if (low_j > -1){
                if (d_visited[low_j] == 0){
                    d_arglow[low_j] = tid;
                    d_classes[tid] = -1;
                    d_clear[low_j] = 1;
                }else{
                    d_ceil_cdim[cdim_pos] = low_j > dim_ceil ? low_j : dim_ceil;
                }
                d_visited[low_j] = 1;
            }
            if (d_dims_order_next[j] == -1){
                iterate = 0;
            }else{
                j = d_dims_order_next[j];
            }
        }
    }
}

void __global__ phase_i(int *d_dims, int *d_dims_order, int *d_low, int *d_arglow, int *d_classes, int *d_clear, int *d_visited, int *d_ceil_cdim, int *d_next_cdim, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int dim_j = d_dims[tid]; // -1, 0, 1, ..., complex_dim
        int cdim_pos = dim_j + 1;
        int j_ord = d_dims_order[tid]; // 0, 1, ...
        int dim_ceil = d_ceil_cdim[cdim_pos]; // initialized at 0
        int low_j = d_low[tid];
        // set lock
        int curr = -1;
        //printf("Entering do-while %d, cdim_pos=%d, j_ord = %d\n", tid, cdim_pos, j_ord);
        do{
            curr = atomicCAS(d_next_cdim+cdim_pos, j_ord, -1);
        } while(curr != j_ord);
        //do {} while(atomicCAS(d_locks_cdim+cdim_pos, j_ord, -1) != j_ord);
        printf("j_ord=%d, ", j_ord);
        // do stuff
        if (low_j > -1){
            if (d_visited[low_j] == 0){
                d_arglow[low_j] = tid;
                d_classes[tid] = -1;
                d_clear[low_j] = 1;
            }else{
                d_ceil_cdim[cdim_pos] = low_j > dim_ceil ? low_j : dim_ceil;
            }
            d_visited[low_j] = 1;
        }
        __threadfence();
        // free lock
        d_next_cdim[cdim_pos] = curr+1;
    }
}

void __global__ phase_ii(int *d_low, int *d_beta, int *d_classes, int *d_arglow, int *d_rows_mp, int *d_aux_mp, int m, int p){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int low_j = d_low[tid];
        int j = tid;
        if (d_arglow[low_j] > -1){
            int pivot = d_arglow[low_j];
            if (pivot < j){
                left_to_right_device(pivot, j, d_rows_mp, d_aux_mp, d_low, m, p);
                // alpha_beta_check 
                low_j = d_low[tid];
                if (low_j > -1){
                    if (d_beta[low_j] == tid){
                        // is lowstar, do a twist clearing
                        d_arglow[low_j] = tid;
                        d_classes[tid] = -1;
                        clear_column(low_j, d_rows_mp, p);
                    }
                }else{
                    d_classes[tid] = 1;
                }
            }
        }
    }
}

void __global__ set_unmarked(int *d_classes, int *d_low, int *d_arglow, int *d_rows_mp, int m, int p){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_classes[tid] == 0){
            if (d_low[tid] > -1){
                d_arglow[tid] = tid;
                d_classes[tid] = -1;
                clear_column(tid, d_rows_mp, p);
            }
        }else{
            d_classes[tid] = 2; 
        }
    }
}

inline void compute_simplex_dimensions(int *d_dims, int *d_dims_order, int *p_complex_dimension, int *d_rows_mp, int m, int p, dim3 numBlocks_m, dim3 threadsPerBlock_m){
    // d_dims
    get_simplex_dimensions<<<numBlocks_m, threadsPerBlock_m>>>(d_dims, d_rows_mp, m, p);
    // d_complex_dim
    thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(d_dims);
    p_complex_dimension[0] = *(thrust::max_element(dev_ptr, dev_ptr + m));
    // d_dims_order
}

inline void compute_simplex_dimensions_h(int *h_rows, int *h_cols, int m, int p, int nnz, int *d_dims, int *d_dims_order, int *d_dims_order_next, int *p_complex_dimension){
    // This one we compute on the host for the moment
    int *h_dims = (int*)malloc( sizeof(int) * m );
    int *h_dims_order = (int*)malloc( sizeof(int) * m );
    // Get simplex dimensions
    for (int i = 0; i < m; i++)
        h_dims[i] = -1;
    for (int i = 0; i < nnz; i++)
        h_dims[h_cols[i]] += 1;

    int complex_dim = -1;
    for (int i = 0; i < m; i++)
        complex_dim = h_dims[i] > complex_dim ? h_dims[i] : complex_dim;
    *p_complex_dimension = complex_dim;

    // Dimensions are {-1, 0, 1, ..., complex_dim}
    int cdim = complex_dim + 2;
    int *h_dims_order_aux = (int*)malloc( sizeof(int) * cdim );
    int *h_dims_order_next = (int*)malloc( sizeof(int) * m );
    int *h_past_cdim = (int*)malloc( sizeof(int) * cdim );

    for (int i = 0; i < cdim; i++)
        h_dims_order_aux[i] = 0;
    for (int i = 0; i < m; i++)
        h_dims_order_next[i] = -1;
    for (int i = 0; i < cdim; i++)
        h_past_cdim[i] = -1;
    int cdim_pos;
    for (int i = 0; i < m; i++){
        cdim_pos = h_dims[i]+1;
        h_dims_order[i] = h_dims_order_aux[cdim_pos];
        h_dims_order_aux[cdim_pos] += 1;
        if (h_past_cdim[cdim_pos] > -1){
            h_dims_order_next[h_past_cdim[cdim_pos]] = i;
        }
        h_past_cdim[cdim_pos] = i;
    }
    // Copy to device
    hipMemcpy(d_dims, h_dims, m*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dims_order, h_dims_order, m*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dims_order_next, h_dims_order_next, m*sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // free
    free(h_past_cdim);
    free(h_dims_order_aux);
    free(h_dims_order_next);
    free(h_dims);
    free(h_dims_order);
}

/*
inline void compute_dimension_order(int *d_dims, int *d_dims_order, int *d_last_pos, int cdim, int m, dim3 numBlocks_m, dim3 threadsPerBlock_m){
    fill<<<numBlocks_m, threadsPerBlock_m>>>(d_last_pos, -1, cdim);
    fill<<<numBlocks_m, threadsPerBlock_m>>>(d_dims_order, -1, m);
    int zero = 0;
    int *d_sentinel;
    hipMalloc((void**)&d_sentinel, sizeof(int));
    //hipMemcpyToSymbol(HIP_SYMBOL("d_sentinel"), &zero, sizeof(int));
    hipMemcpy(d_sentinel, &zero, sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    compute_dims_order<<<numBlocks_m, threadsPerBlock_m>>>(d_dims, d_dims_order, d_last_pos, m, d_sentinel);
    fill<<<numBlocks_m, threadsPerBlock_m>>>(d_last_pos, -1, cdim);
    hipFree(d_sentinel);
}
*/

int is_reduced(int *d_aux, int *d_lows, int m, dim3 numBlocks_m, dim3 threadsPerBlock_m){
    int one = 1;
    int is_reduced;
    hipMemcpyToSymbol(HIP_SYMBOL(d_is_reduced), &one, sizeof(int));
    zero_vector_int<<<numBlocks_m, threadsPerBlock_m>>>(d_aux, m);
    matrix_is_reduced<<<numBlocks_m, threadsPerBlock_m>>>(d_lows, d_aux, m);
    hipMemcpyFromSymbol(&is_reduced, HIP_SYMBOL(d_is_reduced), sizeof(int));
    return is_reduced;
}

inline void create_beta_h(int *h_beta, int *h_rows, int *h_cols, int m, int nnz){
    int *h_visited = (int*)malloc( sizeof(int) * m );
    for(int i=0; i<m; i++) h_visited[i] = 0;
    for(int i=0; i<m; i++) h_beta[i] = -1;
    for(int l=0; l<nnz; l++)
        if (h_visited[h_rows[l]] == 0){
            h_beta[h_cols[l]] = h_beta[h_cols[l]] > h_rows[l] ? h_beta[h_cols[l]] : h_rows[l];
            h_visited[h_rows[l]] = 1;
        }
    free(h_visited);
}

inline void create_beta(int *d_beta, int *h_rows, int *h_cols, int m, int nnz){
    int *h_beta = (int*)malloc( sizeof(int) * m );
    create_beta_h(h_beta, h_rows, h_cols, m, nnz);
    hipMemcpy(d_beta, h_beta, m*sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    free(h_beta);
}


