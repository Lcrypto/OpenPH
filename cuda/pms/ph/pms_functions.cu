#include "hip/hip_runtime.h"

__device__ int is_reduced = 1;
void __global__ matrix_is_reduced(int *d_lows, int *d_aux, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int low_j = d_lows[tid];
        if (low_j > -1){
            atomicAdd(d_aux+low_j, 1);
            if (d_aux[low_j] > 1)
                is_reduced = 0;
        }
    }
} 

void __global__ compute_dims_order(int *d_dims, int *d_dims_order, int *d_last_pos, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int dim_j = d_dims[tid];
        if (tid == 0){
            int pos = d_last_pos[dim_j]+1;
            d_dims_order[tid] = pos;
            d_last_pos[dim_j] = pos;
        }else{
            do {} while (d_dims_order[tid-1] == -1);
            int pos = d_last_pos[dim_j]+1;
            d_dims_order[tid] = pos;
            d_last_pos[dim_j] = pos;
        }
    }
}

void __global__ alpha_beta_reduce(int *d_lows, int *d_beta, int *d_classes, int *d_rows_mp, int *d_arglow, int *d_lowstar, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int alpha = d_lows[tid];
        int beta  = d_beta[tid];
        if (alpha == beta && beta > -1){
            // tid is "negative"
            d_classes[tid] = -1;
            int pos_pair = d_beta[tid];
            clear_column(pos_pair, d_rows_mp, p);
            d_arglow[pos_pair] = beta;
            d_lowstar[pos_pair] = -1;
            d_lowstar[beta] = pos_pair;
        }
    }
}

void __global__ count_simplices_dim(int *d_dim_count, int *d_dims){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_dims[tid] > -1){
            atomicAdd(d_dim_count[d_dims[tid]], 1);
        }
    }
}

void __global__ phase_i(int *d_ceilings, int *d_dims, int *d_dims_order, int *d_low, int *d_arglow, int *d_classes, int *d_clear, int *d_visited, int *d_ceil_cdim, int *d_locks_cdim, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int dim_j = d_dims[tid];
        int j_ord = d_dims_order[tid]; // 0, 1, ...
        int dim_ceil = d_ceil_cdim[dim_j];
        // set lock
        do {} while(atomicCAS(d_locks_cdim[dim_j], j_ord, -1) == j_ord);
        // do stuff
        if (low_j > -1){
            if (d_visited[low_j] == 0){
                d_arglow[low_j] = tid;
                d_classes[tid] = -1;
                d_clear[low_j] = 1;
                d_ceilings[tid] = dim_ceil;
            }else{
                d_ceilings[tid] = dim_ceil;
                d_ceil_cdim[dim_j] = low_j > dim_ceil ? low_j : dim_ceil;
            }
        }
        // free lock
        //d_lock = tid+1;
        d_locks_cdim[dim_j] = j_ord + 1;
    }
}

void __global__ phase_ii(int *d_low, int *d_arglow, int *d_rows_mp, int *d_aux_mp, int m, int p){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int low_j = d_low[tid];
        if (d_arglow[low_j] > -1){
            int pivot = d_arglow[low_j];
            if (pivot < j){
                left_to_right_device(pivot, j, d_rows_mp, d_aux_mp, d_low, m, p);
                d_updated[tid] = 1;
                // alpha_beta_check 
                low_j = d_low[tid];
                if (low_j > -1){
                    if (d_beta[low_j] == tid){
                        // is lowstar, do a twist clearing
                        d_arglow[low_j] = tid;
                        d_classes[tid] = -1;
                        clear_column(low_j, d_rows_mp, p);
                    }
                }else{
                    d_classes[tid] = 1;
                }
            }
        }
    }
}

void __global__ set_unmarked(int *d_classes, int *d_low, int *d_arglow, int *d_rows_mp, int m, int p){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_classes[tid] == 0)
            if (d_low[tid] > -1){
                d_arglow[tid] = tid;
                d_classes[tid] = -1;
                clear_column(tid, d_rows_mp, p);
            }
        }else{
            d_classes[tid] = 2; 
        }
    }
}

inline void compute_simplex_dimensions(int *d_dims, int *d_dims_order, int *p_complex_dimension, int *d_rows_mp, int m, int p, dim3 numBlocks_m, dim3 threadsPerBlock_m){
    // d_dims
    get_simplex_dimensions<<<numBlocks_m, threadsPerBlock_m>>>(d_dims, d_rows_mp, m, p);
    // d_complex_dim
    thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(d_dims);
    thrust::device_ptr<int> max_ptr = thrust::max_element(dev_ptr, dev_ptr + m);
    *p_complex_dimension = max_ptr[0];
}

inline void compute_dimension_order(int *d_dims, int *d_dims_order, int *d_last_pos, int cdim, int m, dim3 numBlocks_m, dim3 threadsPerBlock_m){
    fill<<<numBlocks_m, threadsPerBlock_m>>>(d_last_pos, -1, cdim);
    fill<<<numBlocks_m, threadsPerBlock_m>>>(d_dims_order, -1, m);
    compute_dims_order<<<numBlocks_m, threadsPerBlock_m>>>(d_dims, d_dims_order, d_last_pos, m);
    fill<<<numBlocks_m, threadsPerBlock_m>>>(d_last_pos, -1, cdim);
}

int is_reduced(int *d_aux, int *d_lows, int m, dim3 numBlocks_m, dim3 threadsPerBlock_m){
    int one = 1;
    int is_reduced;
    hipMemcpyToSymbol(HIP_SYMBOL(d_is_reduced), &one, sizeof(int));
    zero_vector_int<<<numBlocks_m, threadsPerBlock_m>>>(d_aux, m);
    matrix_is_reduced<<<numBlocks_m, threadsPerBlock_m>>>(d_lows, d_aux, m);
    hipMemcpyFromSymbol(&is_reduced, HIP_SYMBOL(d_is_reduced), sizeof(int));
    return is_reduced;
}

inline void create_beta(int *d_beta, int *h_rows, int *h_cols, int m, int nnz){
    int *h_beta;
    h_beta = (int*)malloc( sizeof(int) * m );
    create_beta_h(h_beta, h_rows, h_cols, m, nnz);
    hipMemcpy(d_beta, h_beta, m*sizeof(int), hipMemcpyHostToDevice);
    free(h_beta);
}

in

