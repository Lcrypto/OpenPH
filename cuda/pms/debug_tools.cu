
#include <hip/hip_runtime.h>

inline void h2d(int *d_v, int j, int v){
    hipMemcpy(d_v+j, &v, sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return;
}

int d2h(int *d_v, int j){
    int v_j = 0;
    hipMemcpy(&v_j, d_v+j, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return v_j;
}

inline void printvec_float(float *d_v, int m, char* s){
    float val = 0;
    printf("%s\n",s);
    for (int i = 0; i < m; i++){
        hipMemcpy(&val, d_v+i, sizeof(float), hipMemcpyDeviceToHost);
        printf("[%d: %5.6f], ", i, val);
    }
    printf("\n\n");
}

inline void printvec(int *d_v, int m, char* s){
    int val = 0;
    printf("%s\n",s);
    for (int i = 0; i < m; i++){
        hipMemcpy(&val, d_v+i, sizeof(int), hipMemcpyDeviceToHost);
        printf("[%d: %d], ", i, val);
    }
    printf("\n\n");
}

inline void print_matrix_cols_mp(int *d_rows_mp, int m, int p){
    int val = 0;
    for (int j = 0; j < m; j++){
        printf("col (%d): ", j);
        for (int i = 0; i < p; i++){
            hipMemcpy(&val, d_rows_mp+(j*p+i), sizeof(int), hipMemcpyDeviceToHost);
            printf("%d, ", val);
        } 
        printf("\n");
    }
}

inline void print_matrix_cols(int *d_rows, int *d_cols, int nnz){
    int col = -1;
    int val = 0;
    for (int idx = 0; idx < nnz; idx++){
        hipMemcpy(&val, d_cols+idx, sizeof(int), hipMemcpyDeviceToHost);
        if (val != col){
            printf("\n");
            printf("col (%d): ", val);
            col = val;
        }
        hipMemcpy(&val, d_rows+idx, sizeof(int), hipMemcpyDeviceToHost);
        printf("%d, ", val);
        hipMemcpy(&val, d_cols+idx, sizeof(int), hipMemcpyDeviceToHost);
    } 
    printf("\n");
}
