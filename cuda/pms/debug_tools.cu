
#include <hip/hip_runtime.h>

inline void printvec(int *d_v, int m, char* s){
    int val = 0;
    printf("%s\n",s);
    for (int i = 0; i < m; i++){
        hipMemcpy(&val, d_v+i, sizeof(int), hipMemcpyDeviceToHost);
        printf("[%d: %d], ", i, val);
    }
    printf("\n\n");
}

inline void print_matrix_cols_mp(int *d_rows_mp, int m, int p){
    int val = 0;
    for (int j = 0; j < m; j++){
        printf("col (%d): ", j);
        for (int i = 0; i < p; i++){
            hipMemcpy(&val, d_rows_mp+(j*p+i), sizeof(int), hipMemcpyDeviceToHost);
            printf("%d, ", val);
        } 
        printf("\n");
    }
}

inline void print_matrix_cols(int *d_rows, int *d_cols, int nnz){
    int col = -1;
    int val = 0;
    for (int idx = 0; idx < nnz; idx++){
        hipMemcpy(&val, d_cols+idx, sizeof(int), hipMemcpyDeviceToHost);
        if (val != col){
            printf("\n");
            printf("col (%d): ", val);
            col = val;
        }
        hipMemcpy(&val, d_rows+idx, sizeof(int), hipMemcpyDeviceToHost);
        printf("%d, ", val);
        hipMemcpy(&val, d_cols+idx, sizeof(int), hipMemcpyDeviceToHost);
    } 
    printf("\n");
}
