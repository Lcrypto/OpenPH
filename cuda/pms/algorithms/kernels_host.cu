
#include <hip/hip_runtime.h>

inline void left_to_right_host(int j0, int j1, int *h_rows_mp, int *h_aux_mp, int *h_low, int m, int p){
    // Compute symmetric difference of supp(j0) and supp(j1) and store in d_aux
    // If rows are initially sorted, this returns a sorted list
    int idx0 = j0*p; 
    int idx1 = j1*p; 
    int idx0_MAX = (j0+1)*p; 
    int idx1_MAX = (j1+1)*p; 
    int idx = idx1;
    bool idx0_ok = h_rows_mp[idx0] != -1 && idx0 < idx0_MAX;
    bool idx1_ok = h_rows_mp[idx1] != -1 && idx1 < idx1_MAX;
    while (idx0_ok || idx1_ok){
        if (idx0_ok && idx1_ok){
            if (h_rows_mp[idx0] < h_rows_mp[idx1]){
                h_aux_mp[idx++] = h_rows_mp[idx0++];
            }else if (h_rows_mp[idx1] < h_rows_mp[idx0]){
                h_aux_mp[idx++] = h_rows_mp[idx1++];
            }else{
                idx0++;
                idx1++;
                if (idx0 == idx0_MAX-1 || idx1 == idx1_MAX-1)
                    printf("WARNING: Column reaching memalloc limit\n");
            }
        }else{
            if (idx0_ok){
                h_aux_mp[idx++] = h_rows_mp[idx0++];
            }
            if (idx1_ok){
                h_aux_mp[idx++] = h_rows_mp[idx1++];
            }
        }
        idx0_ok = h_rows_mp[idx0] != -1 && idx0 < idx0_MAX;
        idx1_ok = h_rows_mp[idx1] != -1 && idx1 < idx1_MAX;
    }
    int low_j1 = -1;
    // At least one value was written in d_aux_mp
    for (idx1 = j1*p; idx1 < idx1_MAX; idx1++){
        h_rows_mp[idx1] = h_aux_mp[idx1];
        h_aux_mp[idx1] = -1;
        if (h_rows_mp[idx1] > -1)
            low_j1 = h_rows_mp[idx1];
    }
    h_low[j1] = low_j1;
}

inline void clear_column_host(int j, int *h_rows_mp, int p){
    int idx = j*p; 
    int idx_MAX = (j+1)*p; 
    while (idx < idx_MAX && h_rows_mp[idx] != -1){
        h_rows_mp[idx++] = -1;
    }
}

inline void reduce_col_host(int j, int *h_rows_mp, int *h_aux_mp, int *h_low, int *h_arglow, int m, int p){
    for (int tid=0; tid<m; tid++){
        int j0 = -1;
        int low_j = h_low[j]; // low_j = -1, 0, 1, ..., m-1
        while (low_j > -1 && h_arglow[low_j] != -1){
            j0 = h_arglow[low_j];
            left_to_right_host(j0, j, h_rows_mp, h_aux_mp, h_low, m, p);
            low_j = h_low[j];
        }
        low_j = h_low[j];
        if (low_j > -1){
            h_arglow[low_j] = j;
        }
    }
}

inline void update_classes_host(int *h_classes, int *h_low, int *h_arglow, int m){
    for (int tid=0; tid<m; tid++){
        if (h_arglow[tid] > -1){
            h_classes[h_arglow[tid]] = -1;
            h_classes[tid] = 1;
        }
    }
}

inline void ess_hat_host(int *h_essential_hat, int *h_low, int *h_arglow, int m){
    for (int j=0; j<m; j++){
        if (h_low[j] > -1){
            h_essential_hat[h_low[j]] = 0;
        }
        if (h_arglow[j] > -1){
            h_essential_hat[h_arglow[j]] = 0;
            h_essential_hat[j] = 0;
        }
    }
}
