#include "hip/hip_runtime.h"

// -----------------------
// Check if matrix is reduced
// -----------------------

__device__ int d_is_reduced = 1;
void __global__ matrix_is_reduced(int *d_lows, int *d_aux, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        int low_j = d_lows[tid];
        if (low_j > -1){
            atomicAdd(d_aux+low_j, 1);
            if (d_aux[low_j] > 1)
                d_is_reduced = 0;
        }
    }
} 

int is_reduced(int *d_aux, int *d_lows, int m, dim3 numBlocks_m, dim3 threadsPerBlock_m){
    int one = 1;
    int is_reduced;
    hipMemcpyToSymbol(HIP_SYMBOL(d_is_reduced), &one, sizeof(int));
    fill<<<numBlocks_m, threadsPerBlock_m>>>(d_aux, 0, m);
    matrix_is_reduced<<<numBlocks_m, threadsPerBlock_m>>>(d_lows, d_aux, m);
    hipMemcpyFromSymbol(&is_reduced, HIP_SYMBOL(d_is_reduced), sizeof(int));
    return is_reduced;
}

// -----------------------
// Phase 0
// -----------------------

void __global__ mark_pivots_and_clear(int *d_low, int *d_beta, int *d_classes, int *d_rows_mp, int *d_arglow, int m, int p){
    int j = threadIdx.x + blockDim.x*blockIdx.x;
    if (j < m){
        int low_j = d_low[j]; 
        int beta_j = d_beta[j];
        // Check if is pivot
        if (low_j == beta_j && beta_j > -1){
            // j is "negative"
            d_classes[j] = -1;
            // low_j is positive
            clear_column(low_j, d_rows_mp, p);
            d_low[low_j] = -1;
            d_classes[low_j] = 1;
            // Record j as pivot
            d_arglow[low_j] = j;
        }
    }
}

void __global__ transverse_dimensions(int *d_dims, int *d_dims_order, int *d_dims_order_next, int *d_dims_order_start, int *d_low, int *d_arglow, int *d_classes, int *d_clear, int *d_visited, int *d_ceil_cdim, int cdim){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < cdim){
        int dim_j; // -1, 0, 1, ..., complex_dim
        int cdim_pos;
        int dim_ceil; // initialized at -1 
        int low_j;
        int j = d_dims_order_start[tid];
        int iterate = 1;
        while (iterate && j > -1){
            dim_j = d_dims[j];
            cdim_pos = dim_j + 1; 
            dim_ceil = d_ceil_cdim[cdim_pos];
            low_j = d_low[j];
            if (low_j > -1){
                if (d_visited[low_j] == 0){
                    if (d_classes[j] == 0 && low_j > dim_ceil){
                        d_arglow[low_j] = j;
                        d_classes[j] = -1;
                        d_clear[low_j] = 1;
                    }
                }else{
                    d_ceil_cdim[cdim_pos] = low_j > dim_ceil ? low_j : dim_ceil;
                }
                d_visited[low_j] = 1;
            }
            // Iterator
            if (d_dims_order_next[j] == -1){
                iterate = 0;
            }else{
                j = d_dims_order_next[j];
            }
        }
    }
}

void __global__ phase_ii(int *d_low, int *d_left, int *d_classes, int *d_arglow, int *d_rows_mp, int *d_aux_mp, int m, int p){
    int j = threadIdx.x + blockDim.x*blockIdx.x;
    if (j < m){
        int low_j = d_low[j];
        int pivot = d_arglow[low_j];
        if (-1 < pivot && pivot < j && d_classes[j] == 0){
            left_to_right(pivot, j, d_rows_mp, d_aux_mp, d_low, m, p);
            // alpha_beta_check 
            low_j = d_low[j];
            if (low_j > -1){
                if (d_left[low_j] == j){
                    // is lowstar, do a twist clearing
                    d_arglow[low_j] = j;
                    d_classes[j] = -1;
                    clear_column(low_j, d_rows_mp, p);
                    d_classes[low_j] = 1;
                }
            }else{
                d_classes[j] = 1;
            }
        }
    }
}

void __global__ set_unmarked(int *d_classes, int *d_low, int *d_arglow, int *d_rows_mp, int m, int p){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_classes[tid] == 0){
            if (d_low[tid] > -1){
                d_arglow[tid] = tid;
                d_classes[tid] = -1;
                clear_column(tid, d_rows_mp, p);
            }
        }else{
            d_classes[tid] = 2; 
        }
    }
}

void __global__ clear_positives(int *d_clear, int *d_low, int *d_classes, int *d_rows_mp, int m, int p){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_clear[tid] == 1){
            d_low[tid] = -1;
            d_classes[tid] = 1;
            clear_column(tid, d_rows_mp, p);
        }
    }
}

