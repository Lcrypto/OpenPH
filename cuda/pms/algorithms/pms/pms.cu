inline void pms(int *d_low, int *d_arglow, int *d_classes, int *d_ess,
        int *d_rows_mp, const int m, const int p,
        int *d_dims, int *d_dims_order, int *d_dims_order_next, int *d_dims_order_start,
        int complex_dimension, int *d_left, int *d_beta, int *d_aux_mp,
        int *d_low_true, int *d_ess_true, float *d_float_m,
        float *error_lone, float *error_linf, float *error_redu, float *error_ess,
        float *time_track, int *p_iter,
        dim3 NBm, dim3 TPBm, dim3 NBcdim, dim3 TPBcdim){
    //  d_pivots[j] = 1  <=> d_arglow[d_low[j]] = j 

    // time
    float time = 0.0;

    // iter and trackers
    track(0, m, d_low, d_ess, d_classes,
            d_low_true, d_ess_true, d_float_m,
            error_lone, error_linf, error_redu,
            error_ess, time_track, time, NBm, TPBm);

    // Auxiliary variables
    int *d_aux;
    int *d_is_positive;
    hipMalloc((void**)&d_aux, m * sizeof(int));
    hipMalloc((void**)&d_is_positive, m * sizeof(int));

    // -----------------------
    // Do some pre-processing work
    // -----------------------

    // Compute simplex dimensions (on device)
    // Get maximum dimension 
    int cdim = complex_dimension + 2; // -1, 0, 1, 2, ..., complex_dim
    int *d_aux_cdim;    // Auxiliary vector of size cdim 
    hipMalloc((void**)&d_aux_cdim, cdim * sizeof(int));

    // -----------------------
    // Phase 0
    // -----------------------

    // Mark pivots and clear corresponding positives
    mark_pivots_and_clear<<<NBm, TPBm>>>(d_low, d_beta, 
            d_classes, d_rows_mp, d_arglow, m, p);
    hipDeviceSynchronize();

    int converged = is_reduced(d_aux, d_low, m, NBm, TPBm);
    int iter = 1;

    while (! converged ){

        // TIC
        hipEvent_t start, stop;
        tic(&start, &stop);

        // -----------------------
        // Main iteration : Phase I 
        // -----------------------

        fill<<<NBm, TPBm>>>(d_aux, 0, m);
        fill<<<NBm, TPBm>>>(d_aux_cdim, -1, cdim); // d_ceil
        fill<<<NBm, TPBm>>>(d_is_positive, 0, m);
        hipDeviceSynchronize();

        transverse_dimensions<<<NBcdim, TPBcdim>>>(d_dims, 
                d_dims_order, d_dims_order_next, d_dims_order_start, 
                d_low, d_arglow, d_classes, d_is_positive,   
                d_aux, d_aux_cdim, cdim);
        hipDeviceSynchronize();

        clear_positives<<<NBm, TPBm>>>(d_is_positive, 
                d_low, d_classes, d_rows_mp, m, p);
        hipDeviceSynchronize();

        // -----------------------
        // Main iteration : Phase II 
        // -----------------------

        fill<<<NBm, TPBm>>>(d_is_positive, 0, m);
        hipDeviceSynchronize();

        phase_ii<<<NBm, TPBm>>>(d_low, d_left, d_classes, 
                d_is_positive, d_arglow, d_rows_mp, d_aux_mp, m, p);
        hipDeviceSynchronize();

        clear_positives<<<NBm, TPBm>>>(d_is_positive, 
                d_low, d_classes, d_rows_mp, m, p);
        hipDeviceSynchronize();

        // Check again if its reduced
        converged = is_reduced(d_aux, d_low, m, NBm, TPBm);

        // update classes (Not necessary for algo to work)
        update_classes<<<NBm, TPBm>>>(d_classes, d_low, d_arglow, m);
        hipDeviceSynchronize();

        // Essential estimation
        ess_hat<<<NBm, TPBm>>>(d_ess, d_low, d_arglow, m);
        hipDeviceSynchronize();

        // TOC
        toc(start, stop, &time);

        // meausre progress
        track(iter, m, d_low, d_ess, d_classes,
                d_low_true, d_ess_true, d_float_m,
                error_lone, error_linf, error_redu,
                error_ess, time_track, time, NBm, TPBm);

        // iter
        iter++;

    }

    set_unmarked<<<NBm, TPBm>>>(d_classes, d_low, d_arglow, d_rows_mp, m, p);
    hipDeviceSynchronize();

    p_iter[0] = iter;

    hipFree(d_aux_cdim);
    hipFree(d_aux);
    hipFree(d_is_positive);

}

