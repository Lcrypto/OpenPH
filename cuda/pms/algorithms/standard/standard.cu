inline void standard(int *d_low, int *d_arglow, int *d_classes, 
        int *d_ess, int *d_rows_mp, const int m, const int p, 
        int *d_aux_mp, int *d_low_true, int *d_ess_true, float *error_lone, 
        float *error_linf, float *error_redu, float *error_ess, float *time_track,
        int *p_iter, dim3 NBm, dim3 TPBm){

    int iter = 1;
    for(int j = 0; j < m; j++){

        // Create timing
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        // Reduce column
        reduce_col<<<NBm, TPBm>>>(j, d_rows_mp, d_aux_mp, d_low, d_arglow, m, p);
        hipDeviceSynchronize();

        update_classes<<<NBm, TPBm>>>(d_classes, d_low, d_arglow, m);
        hipDeviceSynchronize();

        // end timing
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        // Essential estimation
        ess_hat<<<NBm, TPBm>>>(d_ess, d_low, d_arglow, m);
        hipDeviceSynchronize();

        // iter and trackers
        track(iter, m, d_low, d_classes, d_low_true, d_ess_true, error_lone, error_linf, error_redu, error_ess, time_track, time, NBm, TPBm);

        iter++;

    }
    p_iter[0] = iter;

}
