#include "hip/hip_runtime.h"
inline void standard(int *h_low, int *h_arglow, int *h_classes, 
        int *h_ess, int *h_rows_mp, const int m, const int p, 
        int *h_aux_mp, int *h_low_true, int *h_ess_true, 
        float *h_float_m, float *error_lone,
        float *error_linf, float *error_redu, float *error_ess, 
        float *time_track, int *p_iter){

    // time
    float time = 0.0;

    // iter and trackers
    track_host(0, m, h_low, h_ess, h_classes, 
            h_low_true, h_ess_true, h_float_m, 
            error_lone, error_linf, error_redu, 
            error_ess, time_track, time);

    int iter = 1;
    for(int j = 0; j < m; j++){

        // TIC
        //clock_t tic = clock();
        hipEvent_t start, stop;
        tic(&start, &stop);

        // Work on column "j"
        reduce_col_host(j, h_rows_mp, h_aux_mp, h_low, h_arglow, m, p, h_ess);

        // Update classes host
        if (h_low[j] > -1){
            h_classes[j] = -1;
            h_classes[h_low[j]] = 1;
        }else{
            h_classes[j] = 1;
        }

        // Essential estimation
        if (h_low[j] > -1){
            h_ess[j] = 0;
            h_ess[h_low[j]] = 0;
        }

        // TOC
        toc(start, stop, &time);

        // meausre progress
        track_host(iter, m, h_low, h_ess, h_classes, 
                h_low_true, h_ess_true, h_float_m, 
                error_lone, error_linf, error_redu, 
                error_ess, time_track, time);

        // iter
        iter++;

    }
    p_iter[0] = iter;

}
