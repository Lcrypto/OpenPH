#include "hip/hip_runtime.h"

inline void left_to_right_neighbours_host(int h_pivot, int *h_pivots, int *h_neighbours, int *h_rows_mp, int *h_aux_mp, int *h_low, int *h_arglow, int m, int p){
    int l = 0;
    while(h_neighbours[l] != -1){
        int j = h_neighbours[l];
        left_to_right_host(h_pivot, j, h_rows_mp, h_aux_mp, h_low, m, p);
        if ((h_low[j] != -1) && (h_pivots[h_low[j]] == -1 || j < h_pivots[h_low[j]]))
            h_pivots[h_low[j]] = j;
        l++;
    }
}

inline void get_neighbours_position_host(int pivot, int *h_neighbours, int *h_low, int *h_dim_next, int m){
    int pos = 0;
    int col = h_dim_next[pivot];
    while((col != -1)){
        if (h_low[col] == h_low[pivot]){
            h_neighbours[pos] = col;
            pos++;
        }
        col = h_dim_next[col];
    }
}

inline void ph_row(int *h_low, int *h_arglow, int *h_classes,
        int *h_ess, int *h_rows_mp, const int m, const int p,
        int *h_dim, int *h_dim_order, int *h_dim_next, int *h_dim_start,
        int *h_aux_mp, int *h_low_true, int *h_ess_true, float * h_float_m,
        float *error_lone, float *error_linf, float *error_redu,
        float *error_ess, float *time_track, int *p_iter){

    // time
    float time = 0.0;

    // iter and trackers
    track_host(0, m, h_low, h_ess, h_classes,
            h_low_true, h_ess_true, h_float_m,
            error_lone, error_linf, error_redu,
            error_ess, time_track, time);

    // d_is_neighbour
    int *h_neighbours = (int*)malloc( m * sizeof(int) );
    for (int i = 0; i < m; i++) h_neighbours[i] = -1;

    int *h_pivots = (int*)malloc( m * sizeof(int) );
    for (int i = 0; i < m; i++) h_pivots[i] = -1;
    for (int j = 0; j < m; j++)
        if ((h_low[j] != -1) && (h_pivots[h_low[j]] == -1 || j < h_pivots[h_low[j]]))
                h_pivots[h_low[j]] = j;

    int iter = 1;
    for(int i = m-1; i > -1; i--){

        // TIC
        //clock_t tic = clock();
        hipEvent_t start, stop;
        tic(&start, &stop);

        // Mark neighbours
        // Contrary to parallel case, we store indices of neighbours
        int pivot = h_pivots[i];
        if (pivot != -1){
            get_neighbours_position_host(pivot, h_neighbours, h_low, h_dim_next, m);
            
            // Reduce neighbours
            left_to_right_neighbours_host(pivot, h_pivots, h_neighbours, h_rows_mp, h_aux_mp, h_low, h_arglow, m, p);

            int l = 0;
            while (l<m && h_neighbours[l] != -1)
                h_neighbours[l++] = -1;
        }

        // update classes (Not necessary for algo to work)
        update_classes_host(h_classes, h_low, h_arglow, m);

        // Essential estimation
        ess_hat_host(h_ess, h_low, h_arglow, m);

        // TOC
        //clock_t toc = clock();
        //time = ((float)((double)(toc - tic) / CLOCKS_PER_SEC)) * 1000;
        toc(start, stop, &time);

        // meausre progress
        track_host(iter, m, h_low, h_ess, h_classes,
                h_low_true, h_ess_true, h_float_m,
                error_lone, error_linf, error_redu,
                error_ess, time_track, time);

        // iter
        iter++;

    } 
    p_iter[0] = iter;

    free(h_neighbours);
    free(h_pivots);

}

