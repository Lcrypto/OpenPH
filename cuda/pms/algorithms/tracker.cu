#include "hip/hip_runtime.h"

__global__ void diff(float *d_res, int *d_vec_1, int *d_vec_2, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        d_res[tid] = ((float) d_vec_1[tid] - d_vec_2[tid]);
    }
} 

__global__ void eq_value(float *d_res, int *d_v, int val, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_v[tid] == val){
            d_res[tid] = 1.0f;
        }else{
            d_res[tid] = 0.0f;
        }
    }
} 

__global__ void eq_vectors(float *d_res, int *d_v1, int *d_v2, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        if (d_v1[tid] == d_v2[tid]){
            d_res[tid] = 1;
        }else{
            d_res[tid] = 0;
        }
    }
} 

__global__ void to_float(float *d_float_m, int *d_v, int m){
    int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid < m){
        float val;
        val = (float) d_v[tid];
        d_float_m[tid] = val;
    }
} 

inline float norm_1(float *d_float_m, int m){
    return hipblasSasum(m, d_float_m, 1);
}

inline float norm_inf(float *d_float_m, int m){
    float ninf;
    int pos = hipblasIsamax(m, d_float_m, 1)-1;
    hipMemcpy(&ninf, d_float_m+pos, sizeof(float), hipMemcpyDeviceToHost);
    return abs(ninf); 
}

inline void track(int iter, int m, 
        int *d_low, int *d_ess, int *d_classes, int *d_low_true, 
        int *d_ess_true, float *d_float_m, float *error_lone, 
        float *error_linf, float *error_redu, 
        float *error_ess, float *time_track, float time,
        dim3 NBm, dim3 TPBm){

    // Time
    time_track[iter] = time;

    // (float) d_low_err = d_low - d_low_true
    diff<<<NBm, TPBm>>>(d_float_m, d_low, d_low_true, m);
    hipDeviceSynchronize();

    // norm_lone(d_low, d_low_true)
    error_lone[iter] = norm_1(d_float_m, m);

    // norm_linf(d_low, d_low_true)
    error_linf[iter] = norm_inf(d_float_m, m);

    // |j : d_classes[j] = 0| (number of unreduced columns)
    eq_value<<<NBm, TPBm>>>(d_float_m, d_classes, 0, m);
    hipDeviceSynchronize();
    error_redu[iter] = hipblasSasum(m, d_float_m, 1) / ((float) m);
    hipDeviceSynchronize();

    // |j : d_ess[j] = d_ess_true[j]|/sum(d_ess_true)
    to_float<<<NBm, TPBm>>>(d_float_m, d_ess_true, m);
    hipDeviceSynchronize();
    float num_ess_true = hipblasSasum(m, d_float_m, 1);

    to_float<<<NBm, TPBm>>>(d_float_m, d_ess, m);
    hipDeviceSynchronize();
    float num_ess_hat = hipblasSasum(m, d_float_m, 1);

    if (num_ess_hat > 0){
        error_ess[iter] = num_ess_true/num_ess_hat;
    }else{
        error_ess[iter] = -1.0;
    }

}

