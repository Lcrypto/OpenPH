#include "hip/hip_runtime.h"

inline int get_max_nnz(int *h_index, int m, int nnz){
    // Get maximum nnz in rows/columns from row/column index.
    // h_index_dim is either h_rows or h_cols

    int *h_nnz_index = (int*)malloc( sizeof(int) * m );

    for(int l=0; l<m; l++)   h_nnz_index[l] = 0;
    for(int l=0; l<nnz; l++) h_nnz_index[h_index[l]] += 1;

    int max_nnz_index = 0;

    for(int l=0; l<m; l++){
        if (h_nnz_index[l] > max_nnz_index)
            max_nnz_index = h_nnz_index[l];
    }

    free(h_nnz_index);

    return max_nnz_index;
} 

inline int assert_col_order(int *h_cols, int m, int nnz){
    int is_col_order = 1;
    for (int l=1; l<nnz; l++)
        if (h_cols[l] < h_cols[l-1]){
            is_col_order = 0; 
            break;
        }
    return is_col_order;
}

inline void set_gpu_device(int gpuNumber, int *p_threads_perblock_m, int *p_threads_perblock_nnz, int *p_threads_perblock_mp, int m, int nnz, int p){

    // gpuNumber
    unsigned int max_threads_per_block;

    hipDeviceProp_t dp;
    hipSetDevice(gpuNumber);
    hipGetDeviceProperties(&dp,gpuNumber);
    max_threads_per_block = dp.maxThreadsPerBlock;

    int devCount;
    hipGetDeviceCount(&devCount);
    if ((gpuNumber >= devCount) && (gpuNumber != 0)){
        cout << "This computer has " << devCount 
        << " gpus and gpuNumber was" << endl << "selected at "
        << gpuNumber << " which is larger than admissible." 
        << endl << "gpuNumber has been reset to 0." << endl; 
        gpuNumber = 0;
    }
    hipSetDevice(gpuNumber);
    hipGetDeviceProperties(&dp,gpuNumber);
    max_threads_per_block = dp.maxThreadsPerBlock;

    int mp = m * p;

    *p_threads_perblock_m   = min(m, max_threads_per_block); 
    *p_threads_perblock_nnz = min(nnz, max_threads_per_block);
    *p_threads_perblock_mp  = min(mp, max_threads_per_block);

}
