#include "hip/hip_runtime.h"

/*

Inputs:

    h_rows_in   (int, nnz)
    h_cols_in   (int, nnz)
    m           (int, 1)
    col_width   (int, 1)
    h_low_true  (int, m)
    nnz         (int, 1), length(h_rows_in)

Outputs:

    h_low       (int, m)
    h_ess       (int, m)
    err_linf    (float, m+1)
    err_lone    (float, m+1)
    err_redu    (float, m+1)
    err_ess     (float, m+1)
    time_track  (float, m+1)
    num_iters   (int, 1)

*/

void openph(char algstr, 
        int *h_rows_in, int *h_cols_in, int m, 
        int col_width, int *h_low_true, int nnz,
        int *h_low, int *h_ess, float *err_linf, 
        float *err_lone, float *err_redu, float *err_ess, 
        float *time_track, int *num_iters){

    // -------------------------------
    // Get p
    // -------------------------------

    int p   = col_width * max_nnz(h_cols_in, m, nnz);
    int mp  = m * p;

    if (assert_col_order(h_cols_in, nnz) == 0){
        printf("WARNING: Matrix incorrect!\n");
    }

    // -------------------------------
    // GPU
    // -------------------------------

    int gpu_number = 3;

    int tpb_m   = 0; // threads per block (m)
    int tpb_nnz = 0; // threads per block (nnz)
    int tpb_mp  = 0; // threads per block (mp)

    set_gpu_device(gpu_number, &tpb_m, &tpb_nnz, &tpb_mp, m, nnz, p);

    dim3 TPBnnz(tpb_nnz);
    dim3 NBnnz(num_blocks(nnz, tpb_nnz));

    dim3 TPBm(tpb_m);
    dim3 NBm(num_blocks(m, tpb_m));

    dim3 TPBmp(tpb_mp);
    dim3 NBmp(num_blocks(mp, tpb_mp));


    // -------------------------------
    // Create data on device
    // -------------------------------

    // d_rows, d_cols
    int *d_rows; 
    hipMalloc((void**)&d_rows, mp * sizeof(int));
    create_rows(h_rows_in, h_cols_in, d_rows, m, p, nnz, NBnnz, TPBnnz, NBmp, TPBmp);

    // device vectors
    int *d_low, *d_arglow, *d_classes, *d_ess;
    int *d_aux_mp;

    hipMalloc((void**)&d_low, m * sizeof(int));
    hipMalloc((void**)&d_arglow, m * sizeof(int));
    hipMalloc((void**)&d_classes, m * sizeof(int));
    hipMalloc((void**)&d_ess, m * sizeof(int));
    hipMalloc((void**)&d_aux_mp, mp * sizeof(int));

    fill<<<NBm, TPBm>>>(d_low, -1, m);
    fill<<<NBm, TPBm>>>(d_arglow, -1, m);
    fill<<<NBm, TPBm>>>(d_classes, 0, m);
    fill<<<NBm, TPBm>>>(d_ess, 1, m);
    fill<<<NBmp,TPBmp>>>(d_aux_mp, -1, mp);
    hipDeviceSynchronize();

    compute_low<<<NBm, TPBm>>>(d_rows, d_low, m, p);
    hipDeviceSynchronize();

    // d_float_m
    float *d_float_m;
    hipMalloc((void**)&d_float_m, m * sizeof(float));

    // d_low_true
    int *d_low_true;
    hipMalloc((void**)&d_low_true, m * sizeof(int));
    hipMemcpy(d_low_true, h_low_true, sizeof(int)*m, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // ... compute norm of d_low_true
    to_float<<<NBm, TPBm>>>(d_float_m, d_low_true, m);
    hipDeviceSynchronize();

    float norm1_low_true = norm_1(d_float_m, m); 
    float norminf_low_true = norm_inf(d_float_m, m); 

    // ... Now shift index down
    indexShiftDown<<<NBm, TPBm>>>(d_low_true, m);
    hipDeviceSynchronize();

    // d_ess_true
    int *d_ess_true;
    hipMalloc((void**)&d_ess_true, m * sizeof(int));

    fill<<<NBm, TPBm>>>(d_ess_true, 1, m);
    hipDeviceSynchronize();

    compute_ess_true<<<NBm, TPBm>>>(d_low_true, d_ess_true, m);
    hipDeviceSynchronize();

    // -------------------------------
    // Get PH vectors
    // -------------------------------

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // -------------------------------
    // Dimensions
    // -------------------------------
    
    // d_dim:           Dim of simplex j (-1, 0, 1, ..., complex_dim)
    // d_dim_order[j]:  Order of simplex j in dimension d_dim[j]
    // d_dim_start[d]:  Position of first simplex in dimension "d" in d_dim
    // d_dim_next[j]:   Simplex of dimension d_dim[j] after "j"

    int *d_dim, *d_dim_order, *d_dim_next, *d_dim_start;
    int complex_dim = -1;

    hipMalloc((void**)&d_dim, m * sizeof(int));
    hipMalloc((void**)&d_dim_order, m * sizeof(int));
    hipMalloc((void**)&d_dim_next, m * sizeof(int));

    compute_simplex_dimensions_h(h_cols_in, m, p, nnz, d_dim, d_dim_order, d_dim_next, &complex_dim);

    int cdim = complex_dim + 2;
    hipMalloc((void**)&d_dim_start, cdim * sizeof(int));

    int threads_perblock_cdim = min(cdim, tpb_m);
    dim3 TPBcdim(threads_perblock_cdim);
    dim3 NBcdim(num_blocks(cdim, threads_perblock_cdim));

    create_dim_start(d_dim, d_dim_order, d_dim_start, cdim, m, NBm, TPBm, NBcdim, TPBcdim);

    // left, beta
    int *d_beta, *d_left;
    hipMalloc((void**)&d_beta, m * sizeof(int));
    hipMalloc((void**)&d_left, m * sizeof(int));
    create_beta(d_beta, d_left, h_rows_in, h_cols_in, m, nnz);

    // -------------------------------
    // Algorithms
    // -------------------------------

    int iter  = 0;
    algorithm_factory(algstr, d_low, d_arglow,
            d_classes, d_ess, d_rows, d_dim,
            d_dim_order, d_dim_next, d_dim_start, 
            d_beta, d_left,
            m, p, complex_dim, d_aux_mp, d_low_true, d_ess_true, d_float_m,
            err_lone, err_linf, err_redu, err_ess,
            time_track, &iter, NBm, TPBm, NBcdim,
            TPBcdim);

    // Record iters to output
    num_iters[0] = iter;

    // scale remaining trackers
    for(int i=0; i<m+1; i++)
        err_lone[i] = err_lone[i]/norm1_low_true;

    for(int i=0; i<m+1; i++)
        err_linf[i] = err_linf[i]/norminf_low_true;

    // matrix: device to host
    indexShiftUp<<<NBm, TPBm>>>(d_low, m); 
    hipDeviceSynchronize();

    hipMemcpy(h_low, d_low, sizeof(int)*m, hipMemcpyDeviceToHost);
    hipMemcpy(h_ess, d_ess, sizeof(int)*m, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // hipFree

    hipFree(d_low);
    hipFree(d_arglow);
    hipFree(d_classes);
    hipFree(d_ess);

    hipFree(d_low_true);
    hipFree(d_ess_true);

    hipFree(d_beta);
    hipFree(d_left);

    hipFree(d_rows);
    hipFree(d_aux_mp);

    hipFree(d_float_m);

    hipFree(d_dim);
    hipFree(d_dim_order);
    hipFree(d_dim_next);
    hipFree(d_dim_start);

    hipDeviceSynchronize();
    cublasShutdown();

    return;

}

