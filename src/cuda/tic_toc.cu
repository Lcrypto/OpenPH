
inline void tic(hipEvent_t *p_start, hipEvent_t *p_stop){
    //hipEvent_t start, stop;
    hipEventCreate(p_start);
    hipEventCreate(p_stop);
    hipEventRecord(p_start[0], 0);
}

inline void toc(hipEvent_t start, hipEvent_t stop, float *p_time){
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(p_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
